#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2015 Open Source Robotics Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */
#include <optix.h>
#include <optixu/optixu_math.h>
#include <optixu/optixu_aabb.h>

rtDeclareVariable(float3, scale, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );
rtDeclareVariable(float2, texCoord, attribute texCoord, );

static __inline__ __device__ bool ReportPotentialIntersect(float t, float3 n,
    float2 uv)
{
  if (rtPotentialIntersection(t))
  {
    shadingNormal = geometricNormal = n;
    texCoord = uv;
    return rtReportIntersection(0);
  }

  return false;
}

RT_PROGRAM void Intersect(int)
{
  float hh = scale.z / 2;
  float radius = scale.x / 2; // TODO: handle scale.y
  float3 origin = ray.origin;
  float3 direction = ray.direction;

  float a = direction.x * direction.x + direction.y * direction.y;
  float b = 2 * (origin.x * direction.x + origin.y * direction.y);
  float c = origin.x * origin.x + origin.y * origin.y - radius * radius;

  float s = b * b - 4 * a * c;

  if (s < 0)
  {
    return;
  }

  float t1 = (-b + sqrt(s)) / (2 * a);
  float t2 = (-b - sqrt(s)) / (2 * a);

  float3 p1 = origin + t1 * direction;
  float3 p2 = origin + t2 * direction;

  float zmin = fminf(p1.z, p2.z);
  float zmax = fmaxf(p1.z, p2.z);

  if (zmin > hh || zmax < -hh)
  {
    return;
  }

  float3 p = (t1 < t2) ? p1 : p2;
  float3 n;
  float2 uv;
  float t;

  if (p.z > hh)
  {
    t = (hh - origin.z) / direction.z;
    p = origin + t * direction;
    n = make_float3(0, 0, 1);

    uv.x = atan2(p.y, p.x) / M_PI;
    uv.y = (p.x * p.x + p.y * p.y) / (radius * radius);
  }
  else if (p.z < -hh)
  {
    t = (-hh - origin.z) / direction.z;
    p = origin + t * direction;
    n = make_float3(0, 0, -1);

    uv.x = atan2(p.y, p.x) / M_PI;
    uv.y = (p.x * p.x + p.y * p.y) / (radius * radius);
  }
  else
  {
    t = fminf(t1, t2);
    n = make_float3(p.x, p.y, 0);
    n = normalize(n);
    uv.x = atan2(p.y, p.x) / M_PI;
    uv.y = p.z;
  }

  ReportPotentialIntersect(t, n, uv);
}

RT_PROGRAM void Bounds(int, float result[6])
{
  float3 ex = scale / 2;
  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->set(-ex, ex);
}
