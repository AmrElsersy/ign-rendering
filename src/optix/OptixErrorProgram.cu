#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>

rtDeclareVariable(float3, color, , );
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtBuffer<float3, 2> buffer;

RT_PROGRAM void Error()
{
  buffer[launchIndex] = color;
}
