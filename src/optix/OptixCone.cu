#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2015 Open Source Robotics Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */
#include <optix.h>
#include <optixu/optixu_math.h>
#include <optixu/optixu_aabb.h>

rtDeclareVariable(float3, scale, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );
rtDeclareVariable(float2, texCoord, attribute texCoord, );

static __inline__ __device__ bool ReportPotentialIntersect(float _t,
    const float3 &_n, const float2 &_uv)
{
  if (rtPotentialIntersection(_t))
  {
    shadingNormal = geometricNormal = _n;
    texCoord = _uv;
    return rtReportIntersection(0);
  }

  return false;
}

RT_PROGRAM void Intersect(int)
{
  float height = scale.z;
  float radius = scale.x / 2; // TODO: handle scale.y

  float3 direction = ray.direction;
  float3 origin = ray.origin;
  origin.z -= scale.z / 2;

  float k = radius / height;
  float k2 = k * k;

  float a = direction.x * direction.x + direction.y * direction.y - direction.z
    * direction.z * k2;

  float b = 2 * (origin.x * direction.x + origin.y * direction.y - origin.z *
      direction.z * k2);

  float c = origin.x * origin.x + origin.y * origin.y - origin.z * origin.z *
    k2;

  float s = b * b - 4 * a * c;

  if (s < 0)
  {
    return;
  }

  float t1 = (-b + sqrt(s)) / (2 * a);
  float t2 = (-b - sqrt(s)) / (2 * a);

  float3 p1 = origin + t1 * direction;
  float3 p2 = origin + t2 * direction;

  float ymin = fminf(p1.z, p2.z);
  float ymax = fmaxf(p1.z, p2.z);

  if (ymin > 0 || ymax < -height || (ymin < -height && ymax > 0))
  {
    return;
  }

  float3 p = (t1 < t2) ? p1 : p2;
  float3 n;
  float2 uv;
  float t;

  float r2 = radius * radius;

  if (origin.z < -height)
  {
    t = (-height - origin.z) / direction.z;
    float3 p3 = origin + t * direction;

    if (p3.x * p3.x + p3.y * p3.y < r2)
    {
      n = make_float3(0, 0, -1);
      ReportPotentialIntersect(t, n, uv);
      return;
    }

    uv.x = atan2(p.y, p.x) / M_PI;
    uv.y = (p.x * p.x + p.y * p.y) / (radius * radius);
  }

  if (p.z > 0)
  {
    p = (t1 < t2) ? p2 : p1;
    t = (t1 < t2) ? t2 : t1;
    float r = radius * p.z / height;
    n = make_float3(0, 0, r * k) - p;
    n.z = -n.z;
    n = normalize(n);
    uv.x = atan2(p.y, p.x) / M_PI;
    uv.y = p.z;
  }
  else if (p.z < -height)
  {
    t = (-height - origin.z) / direction.z;
    p = origin + t * direction;
    n = make_float3(0, 0, -1);

    if (p.x * p.x + p.y * p.y > r2)
    {
      return;
    }

    uv.x = atan2(p.y, p.x) / M_PI;
    uv.y = (p.x * p.x + p.y * p.y) / (radius * radius);
  }
  else
  {
    t = fminf(t1, t2);
    float r = radius * p.z / height;
    n = make_float3(0, 0, r * k) - p;
    n.z = -n.z;
    n = normalize(n);
    uv.x = atan2(p.y, p.x) / M_PI;
    uv.y = p.z;
  }

  ReportPotentialIntersect(t, n, uv);
}

RT_PROGRAM void Bounds(int, float _result[6])
{
  float3 ex = scale / 2;
  optix::Aabb* aabb = (optix::Aabb*)_result;
  aabb->set(-ex, ex);
}
