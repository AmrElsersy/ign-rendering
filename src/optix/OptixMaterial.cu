#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>
#include "ignition/rendering/optix/OptixRayTypes.hh"
#include "ignition/rendering/optix/OptixLightTypes.hh"

// scene variables
rtDeclareVariable(float3, ambientLightColor, , );
rtDeclareVariable(int, maxReflectionDepth, , );
rtDeclareVariable(float, sceneEpsilon, , );
rtDeclareVariable(rtObject, rootGroup, , );
rtBuffer<OptixDirectionalLightData> directionalLights;
rtBuffer<OptixPointLightData> pointLights;

// material variables
rtDeclareVariable(float3, ambient, , );
rtDeclareVariable(float3, diffuse, , );
rtDeclareVariable(float, reflectivity, , );

// ray variables
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(OptixRadianceRayData, radianceData, rtPayload, );
rtDeclareVariable(OptixShadowRayData, shadowData, rtPayload, );

// intersect variables
rtDeclareVariable(float, hitDist, rtIntersectionDistance, );
rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );

RT_PROGRAM void AnyHit()
{
  shadowData.attenuation = make_float3(0);
  rtTerminateRay();
}

RT_PROGRAM void ClosestHit()
{
  float3 color = ambient * ambientLightColor;
  // color += hitDist * make_float3(0.025, 0.025, 0.025);

  float3 worldGeomNorm = normalize(
      rtTransformNormal(RT_OBJECT_TO_WORLD, geometricNormal));

  float3 worldShadeNorm = normalize(
      rtTransformNormal(RT_OBJECT_TO_WORLD, shadingNormal));

  float3 forwardNormal = faceforward(worldShadeNorm, -ray.direction,
      worldGeomNorm);

  float3 hitPoint = ray.origin + hitDist * ray.direction;

  for (int i = 0; i < directionalLights.size(); ++i)
  {
    OptixDirectionalLightData light = directionalLights[i];
    float3 l = normalize(-light.direction);
    float ndl = dot(forwardNormal, l);

    if (ndl > 0)
    {
      OptixShadowRayData data;
      data.attenuation = make_float3(1);
      optix::Ray shadowRay(hitPoint, l, RT_SHADOW, sceneEpsilon, 1000);
      rtTrace(rootGroup, shadowRay, data);
      float3 attenuation = data.attenuation;

      if (fmaxf(attenuation) > 0)
      {
        // TODO: add light's attenuation
        float4 ld4 = light.common.color.diffuse;
        float3 Lc = make_float3(ld4.x, ld4.y, ld4.z) * attenuation;
        color += diffuse * ndl * Lc;

        float3 H = normalize(l - ray.direction);
        float nDh = dot( forwardNormal, H );

        if(nDh > 0)
        {
          // TODO: include material specular
          // float4 ks4 = light.common.color.specular;
          // float3 Ks = make_float3(ks4.x, ks4.y, ks4.z) * attenuation;
          float3 Ks = make_float3(0.5, 0.5, 0.5);
          float phong_exp = 50;
          color += Ks * Lc * pow(nDh, phong_exp);
        }
      }
    }
  }

  for (int i = 0; i < pointLights.size(); ++i)
  {
    OptixPointLightData light = pointLights[i];
    float3 l = normalize(light.common.position - hitPoint);
    float ndl = dot(forwardNormal, l);

    if (ndl > 0)
    {
      OptixShadowRayData data;
      data.attenuation = make_float3(1);
      float dist = length(light.common.position - hitPoint);
      optix::Ray shadowRay(hitPoint, l, RT_SHADOW, sceneEpsilon, dist);
      rtTrace(rootGroup, shadowRay, data);
      float3 attenuation = data.attenuation;

      if (fmaxf(attenuation) > 0)
      {
        // TODO: add light's attenuation
        float4 ld4 = light.common.color.diffuse;
        float3 Lc = make_float3(ld4.x, ld4.y, ld4.z) * attenuation;
        color += diffuse * ndl * Lc;

        float3 H = normalize(l - ray.direction);
        float nDh = dot( forwardNormal, H );

        if(nDh > 0)
        {
          // TODO: include material specular
          // float4 ks4 = light.common.color.specular;
          // float3 Ks = make_float3(ks4.x, ks4.y, ks4.z) * attenuation;
          float3 Ks = make_float3(0.5, 0.5, 0.5);
          float phong_exp = 50;
          color += Ks * Lc * pow(nDh, phong_exp);
        }
      }
    }
  }

  if (reflectivity > 0 && radianceData.depth < maxReflectionDepth)
  {
    OptixRadianceRayData refData;
    refData.depth = radianceData.depth + 1;
    float3 R = reflect(ray.direction, forwardNormal);
    optix::Ray refRay(hitPoint, R, RT_RADIANCE, sceneEpsilon);
    rtTrace(rootGroup, refRay, refData);
    color += reflectivity * refData.color;
  }

  radianceData.color = color;
}
