#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2015 Open Source Robotics Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */
#include <optix.h>
#include <optix_math.h>
#include "ignition/rendering/optix/OptixRayTypes.hh"
#include "ignition/rendering/optix/OptixLightTypes.hh"

// scene variables
rtDeclareVariable(float3, ambientLightColor, , );
rtDeclareVariable(int, maxReflectionDepth, , );
rtDeclareVariable(int, maxRefractionDepth, , );
rtDeclareVariable(float, importanceCutoff, , );
rtDeclareVariable(float, sceneEpsilon, , );
rtDeclareVariable(rtObject, rootGroup, , );
rtBuffer<OptixDirectionalLightData> directionalLights;
rtBuffer<OptixPointLightData> pointLights;
rtTextureSampler<float4, 2> texSampler;

// material variables
rtDeclareVariable(float3, ambient, , );
rtDeclareVariable(float3, diffuse, , );
rtDeclareVariable(float, reflectivity, , );
rtDeclareVariable(float, transparency, , );

// ray variables
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(OptixRadianceRayData, radianceData, rtPayload, );
rtDeclareVariable(OptixShadowRayData, shadowData, rtPayload, );

// intersect variables
rtDeclareVariable(float, hitDist, rtIntersectionDistance, );
rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );
rtDeclareVariable(float2, texCoord, attribute texCoord, );

static __device__ __inline__ float3 Exp(const float3 &_x)
{
  return make_float3(exp(_x.x), exp(_x.y), exp(_x.z));
}

RT_PROGRAM void AnyHit()
{
  float3 shadowAtten   = diffuse;

  if (transparency > 0)
  {
    float3 worldNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,
          shadingNormal));

    float ndi = fabs(dot(worldNormal, ray.direction));

    shadowData.attenuation *= 1 - fresnel_schlick(ndi, 5, 1 - shadowAtten,
        make_float3(1));

    // float3 ones = make_float3(1);
    // shadowData.attenuation *= 1 - transparency * (ones - ambient) *
    //     (2 - fresnel_schlick(ndi, 5, 1 - shadowAtten, make_float3(1)));

    rtIgnoreIntersection();
  }
  else
  {
    shadowData.attenuation = make_float3(0);
    rtTerminateRay();
  }
}

RT_PROGRAM void ClosestHit()
{
  float  fresnelExp    = 3.0;
  float  fresnelMin    = 0.1;
  float  fresnelMax    = 1.0;
  float  refractIndex  = 1.4;
  float3 refractColor  = diffuse;
  float3 extinctConst  = diffuse;
  float3 cutoffColor   = diffuse;

  float3 color = ambient * ambientLightColor;

  float3 worldGeomNorm = normalize(
      rtTransformNormal(RT_OBJECT_TO_WORLD, geometricNormal));

  float3 worldShadeNorm = normalize(
      rtTransformNormal(RT_OBJECT_TO_WORLD, shadingNormal));

  float3 forwardNormal = faceforward(worldShadeNorm, -ray.direction,
      worldGeomNorm);

  float3 hitPoint = ray.origin + hitDist * ray.direction;

  float reflection = 1.0;
  float3 result = make_float3(0);
  float3 beerAtten = make_float3(1);

  if (transparency > 0)
  { 
    float3 beerAtten = (dot(worldShadeNorm, ray.direction) > 0) ?
      Exp(extinctConst * hitDist) : make_float3(1);

    if (radianceData.depth < maxRefractionDepth)
    {
      float3 t;

      if (refract(t, ray.direction, worldShadeNorm, refractIndex))
      {
        float cosTheta = dot(ray.direction, worldGeomNorm);
        cosTheta = (cosTheta < 0) ? -cosTheta : dot(t, worldShadeNorm);

        reflection = fresnel_schlick(cosTheta, fresnelExp, fresnelMin,
            fresnelMax);

        float importance = radianceData.importance * (1 - reflection) *
          optix::luminance(refractColor * beerAtten);

        if (importance > importanceCutoff)
        {
          optix::Ray ray(hitPoint, t, RT_RADIANCE, sceneEpsilon);
          OptixRadianceRayData refrData;
          refrData.depth = radianceData.depth + 1;
          refrData.importance = importance;
          rtTrace(rootGroup, ray, refrData);
          result += (1 - reflectivity) * refractColor * refrData.color;
        }
        else
        {
          result += (1 - reflectivity) * refractColor * cutoffColor;
        }
      }
    }
  }

  for (int i = 0; i < directionalLights.size(); ++i)
  {
    OptixDirectionalLightData light = directionalLights[i];
    float3 l = normalize(-light.direction);
    float ndl = dot(forwardNormal, l);

    if (ndl > 0)
    {
      OptixShadowRayData data;
      data.attenuation = make_float3(1);
      optix::Ray shadowRay(hitPoint, l, RT_SHADOW, sceneEpsilon, 1000);
      rtTrace(rootGroup, shadowRay, data);
      float3 attenuation = data.attenuation;

      if (fmaxf(attenuation) > 0)
      {
        OptixLightAttenuation att = light.common.atten;
        float attp = 1 - fminf(hitDist, att.range) / att.range;

        float attf = att.constant + attp * att.linear + attp *
          attp * att.quadratic;

        // rtPrintf("1 - (%f / %f) = %f\n", hitDist, att.range, attp);

        // rtPrintf("%f + %f * %f + %f * %f * %f = %f\n", att.constant, attp,
        //     att.linear, attp, attp, att.quadratic, attf);

        float4 ld4 = light.common.color.diffuse;
        float3 Lc = make_float3(ld4.x, ld4.y, ld4.z) * attenuation;
        color += diffuse * ndl * Lc * attf;

        float3 H = normalize(l - ray.direction);
        float nDh = dot( forwardNormal, H );

        if(nDh > 0)
        {
          // TODO: include material specular
          // float4 ks4 = light.common.color.specular;
          // float3 Ks = make_float3(ks4.x, ks4.y, ks4.z) * attenuation;
          float3 Ks = make_float3(0.5, 0.5, 0.5);
          float phong_exp = 50;
          color += Ks * Lc * pow(nDh, phong_exp);
        }
      }
    }
  }

  for (int i = 0; i < pointLights.size(); ++i)
  {
    OptixPointLightData light = pointLights[i];
    float3 l = normalize(light.common.position - hitPoint);
    float ndl = dot(forwardNormal, l);

    if (ndl > 0)
    {
      OptixShadowRayData data;
      data.attenuation = make_float3(1);
      float dist = length(light.common.position - hitPoint);
      optix::Ray shadowRay(hitPoint, l, RT_SHADOW, sceneEpsilon, dist);
      rtTrace(rootGroup, shadowRay, data);
      float3 attenuation = data.attenuation;

      if (fmaxf(attenuation) > 0)
      {
        // TODO: add light's attenuation
        float4 ld4 = light.common.color.diffuse;
        float3 Lc = make_float3(ld4.x, ld4.y, ld4.z) * attenuation;
        color += diffuse * ndl * Lc;

        float3 H = normalize(l - ray.direction);
        float nDh = dot( forwardNormal, H );

        if(nDh > 0)
        {
          // TODO: include material specular
          // float4 ks4 = light.common.color.specular;
          // float3 Ks = make_float3(ks4.x, ks4.y, ks4.z) * attenuation;
          float3 Ks = make_float3(0.5, 0.5, 0.5);
          float phong_exp = 50;
          color += Ks * Lc * pow(nDh, phong_exp);
        }
      }
    }
  }

  if (reflectivity > 0 && radianceData.depth < maxReflectionDepth)
  {
    OptixRadianceRayData refData;
    refData.depth = radianceData.depth + 1;
    float3 R = reflect(ray.direction, forwardNormal);
    optix::Ray refRay(hitPoint, R, RT_RADIANCE, sceneEpsilon);
    rtTrace(rootGroup, refRay, refData);
    color += reflectivity * refData.color;
  }

  const float2 uv = texCoord;
  float3 tcolor = make_float3(tex2D(texSampler, uv.x, uv.y));
  float3 finalColor = color + color * tcolor * tcolor * tcolor;

  radianceData.color = (1 - transparency) * finalColor +
      (transparency * result * beerAtten);
}
