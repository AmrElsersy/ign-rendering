#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>
#include "ignition/rendering/optix/OptixRayTypes.hh"
#include "ignition/rendering/optix/OptixLightTypes.hh"

// scene variables
rtDeclareVariable(float3, ambientLightColor, , );
rtDeclareVariable(int, maxReflectionDepth, , );
rtDeclareVariable(float, sceneEpsilon, , );
rtDeclareVariable(rtObject, rootGroup, , );
rtBuffer<OptixPointLightData> pointLights;

// material variables
rtDeclareVariable(float3, ambient, , );
rtDeclareVariable(float3, diffuse, , );
rtDeclareVariable(float, reflectivity, , );

// ray variables
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(OptixRadianceRayData, radianceData, rtPayload, );
rtDeclareVariable(OptixShadowRayData, shadowData, rtPayload, );

// intersect variables
rtDeclareVariable(float, hitDist, rtIntersectionDistance, );
rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );

RT_PROGRAM void AnyHit()
{
  shadowData.attenuation = make_float3(0);
  rtTerminateRay();
}

RT_PROGRAM void ClosestHit()
{
  float3 color = ambient * ambientLightColor;
  // color += hitDist * make_float3(0.025, 0.025, 0.025);

  float3 worldGeomNorm = normalize(
      rtTransformNormal(RT_OBJECT_TO_WORLD, geometricNormal));

  float3 worldShadeNorm = normalize(
      rtTransformNormal(RT_OBJECT_TO_WORLD, shadingNormal));

  float3 forwardNormal = faceforward(worldShadeNorm, -ray.direction,
      worldGeomNorm);

  float3 hitPoint = ray.origin + hitDist * ray.direction;

  for (int i = 0; i < pointLights.size(); ++i)
  {
    OptixPointLightData light = pointLights[i];
    float3 l = normalize(light.common.position - hitPoint);
    float ndl = dot(forwardNormal, l);

    // int c = pointLights.size();
    // rtPrintf("%d / %d : %f, %f, %f\n", i, c, l.x, l.y, l.z);

    if (ndl > 0)
    {
      OptixShadowRayData data;
      data.attenuation = make_float3(1);
      float dist = length(light.common.position - hitPoint);
      optix::Ray shadowRay(hitPoint, l, RT_SHADOW, sceneEpsilon, dist);
      rtTrace(rootGroup, shadowRay, data);
      float3 attenuation = data.attenuation;

      if (fmaxf(attenuation) > 0)
      {
        // TODO: add light's attenuation
        float4 ld4 = light.common.color.diffuse;
        float3 Lc = make_float3(ld4.x, ld4.y, ld4.z) * attenuation;
        color += diffuse * ndl * Lc;

        float3 H = normalize(l - ray.direction);
        float nDh = dot( forwardNormal, H );

        if(nDh > 0)
        {
          // TODO: include material specular
          // float3 Ks = make_float3(0.8, 0.9, 0.8);
          float4 ks4 = light.common.color.specular;
          float3 Ks = make_float3(ks4.x, ks4.y, ks4.z) * attenuation;
          float phong_exp = 88;
          color += Ks * Lc * pow(nDh, phong_exp);
        }
      }
    }
  }

  if (reflectivity > 0 && radianceData.depth < maxReflectionDepth)
  {
    OptixRadianceRayData refData;
    refData.depth = radianceData.depth + 1;
    float3 R = reflect(ray.direction, forwardNormal);
    optix::Ray refRay(hitPoint, R, RT_RADIANCE, sceneEpsilon);
    rtTrace(rootGroup, refRay, refData);
    color += reflectivity * refData.color;
  }

  radianceData.color = color;
}
