#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math.h>
#include <optixu/optixu_aabb.h>

rtBuffer<float3> vertexBuffer;
rtBuffer<float3> normalBuffer;
rtBuffer<float2> texCoordBuffer;
rtBuffer<int3>   indexBuffer;
rtDeclareVariable(float3, scale, , );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );

RT_PROGRAM void Intersect(int primIndex)
{
  int3 indices = indexBuffer[primIndex];

  float3 p0 = scale * vertexBuffer[ indices.x ];
  float3 p1 = scale * vertexBuffer[ indices.y ];
  float3 p2 = scale * vertexBuffer[ indices.z ];

  float3 n;
  float t, beta, gamma;

  if (intersect_triangle(ray, p0, p1, p2, n, t, beta, gamma))
  {

    if (rtPotentialIntersection(t))
    {
      if (normalBuffer.size() == 0 || indices.x < 0 || indices.y < 0 ||
          indices.z < 0 )
      {
        shadingNormal = normalize(n);
      }
      else
      {
        float3 n0 = normalBuffer[ indices.x ] / scale;
        float3 n1 = normalBuffer[ indices.y ] / scale;
        float3 n2 = normalBuffer[ indices.z ] / scale;

        shadingNormal = normalize(n1 * beta + n2 * gamma + n0 *
          (1.0f - beta - gamma));
      }

      geometricNormal = normalize(n);

      if ( texCoordBuffer.size() == 0 || indices.x < 0 || indices.y < 0 ||
        indices.z < 0 )
      {
        texcoord = make_float3( 0.0f, 0.0f, 0.0f );
      }
      else
      {
        float2 t0 = texCoordBuffer[ indices.x ];
        float2 t1 = texCoordBuffer[ indices.y ];
        float2 t2 = texCoordBuffer[ indices.z ];

        texcoord = make_float3(t1 * beta + t2 * gamma +
            t0 * (1.0f - beta-gamma));
      }

      rtReportIntersection(0);
    }

  }
}

RT_PROGRAM void Bounds(int primIndex, float result[6])
{
  // TODO: precalculate once

  const int3 indices = indexBuffer[primIndex];

  const float3 v0   = vertexBuffer[ indices.x ];
  const float3 v1   = vertexBuffer[ indices.y ];
  const float3 v2   = vertexBuffer[ indices.z ];
  const float  area = length(cross(v1 - v0, v2 - v0));

  optix::Aabb* aabb = (optix::Aabb*)result;

  if(area > 0.0f && !isinf(area))
  {
    aabb->m_min = scale * fminf(fminf(v0, v1), v2 );
    aabb->m_max = scale * fmaxf(fmaxf(v0, v1), v2 );
  }
  else
  {
    aabb->invalidate();
  }
}
