#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>
#include "ignition/rendering/optix/OptixRayTypes.hh"

rtDeclareVariable(float3, color, , );
rtDeclareVariable(OptixRadianceRayData, payload, rtPayload, );

RT_PROGRAM void Miss()
{
  payload.color = color;
}
