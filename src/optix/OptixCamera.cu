#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>
#include "ignition/rendering/optix/OptixRayTypes.hh"

// camera variables
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3,   u, , );
rtDeclareVariable(float3,   v, , );
rtDeclareVariable(float3,   w, , );
rtDeclareVariable(uint,    aa, , );
rtBuffer<float3, 2> buffer;

// current ray variables
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );

// scene variables
rtDeclareVariable(rtObject, rootGroup, , );
rtDeclareVariable(float, sceneEpsilon, , );

RT_PROGRAM void Render()
{
  // setup subpixel data
  float step = 1.0 / aa;
  float2 pixel = make_float2(launchIndex) + (step / 2);
  float2 size  = make_float2(launchDim);
  int pixCount = aa * aa;

  // clear current pixel buffer
  buffer[launchIndex] = make_float3(0, 0, 0);

  float y0 = pixel.y;
  OptixRadianceRayData data;

  // process each subpixel column
  for (int x = 0; x < aa; ++x)
  {
    // process each subpixel row
    for (int y = 0; y < aa; ++y)
    {
      // create new ray for subpixel
      float2 ratio = pixel / size - 0.5;
      float3 direction = normalize(ratio.x * u + ratio.y * v + w);
      optix::Ray ray(eye, direction, RT_RADIANCE, sceneEpsilon);

      // reset ray data
      data.depth = 0;
      data.importance = 1;
      data.color = make_float3(0, 0, 0);

      // cast ray & update buffer
      rtTrace(rootGroup, ray, data);
      buffer[launchIndex] += data.color;

      // increment y
      pixel.y += step;
    }

    // increment x & reset y
    pixel.x += step;
    pixel.y = y0;
  }

  // compute mean of all subpixels
  buffer[launchIndex] /= pixCount;
}
