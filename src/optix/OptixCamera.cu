#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2015 Open Source Robotics Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */
#include <optix.h>
#include <optix_math.h>
#include "ignition/rendering/optix/OptixRayTypes.hh"

// camera variables
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3,   u, , );
rtDeclareVariable(float3,   v, , );
rtDeclareVariable(float3,   w, , );
rtDeclareVariable(uint,    aa, , );
rtBuffer<float3, 2> buffer;

// current ray variables
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );

// scene variables
rtDeclareVariable(rtObject, rootGroup, , );
rtDeclareVariable(float, sceneEpsilon, , );

RT_PROGRAM void Render()
{
  // setup subpixel data
  float step = 1.0 / aa;
  float2 pixel = make_float2(launchIndex) + (step / 2);
  float2 size  = make_float2(launchDim);
  int pixCount = aa * aa;

  // clear current pixel buffer
  buffer[launchIndex] = make_float3(0, 0, 0);

  float y0 = pixel.y;
  OptixRadianceRayData data;

  // process each subpixel column
  for (int x = 0; x < aa; ++x)
  {
    // process each subpixel row
    for (int y = 0; y < aa; ++y)
    {
      // create new ray for subpixel
      float2 ratio = pixel / size - 0.5;
      float3 direction = normalize(ratio.x * u + ratio.y * v + w);
      optix::Ray ray(eye, direction, RT_RADIANCE, sceneEpsilon);

      // reset ray data
      data.depth = 0;
      data.importance = 1;
      data.color = make_float3(0, 0, 0);

      // cast ray & update buffer
      rtTrace(rootGroup, ray, data);
      buffer[launchIndex] += data.color;

      // increment y
      pixel.y += step;
    }

    // increment x & reset y
    pixel.x += step;
    pixel.y = y0;
  }

  // compute mean of all subpixels
  buffer[launchIndex] /= pixCount;
}
